
#include<stdio.h>
#include"scrImagePgmPpmPackage.h"





int main(int argc, const char * argv[])
{
	int height=0, width =0, scaled_height=0,scaled_width=0;
	//Define the scaling ratio	
	float scaling_ratio=0.5;
	unsigned char*data;
	unsigned char*scaled_data,*d_scaled_data;

	char inputStr[] = {"../../Chapter02/02_memory_overview/05_image_scaling/aerosmith-double.pgm"};
	char outputStr[1024] = {"aerosmith-double-scaled.pgm"};

    hipError_t return_value;

    hipArray* cu_array;

    hipChannelFormatKind kind = hipChannelFormatKindUnsigned;
    hipChannelFormatDesc channel_desc = hipCreateChannelDesc(8,0,0,0,kind);

    get_PgmPpmParams(inputStr,&height,&width);
    data  = (unsigned char*)malloc(width*height*sizeof(unsigned char));
    printf("\n Reading image width height and width [%d][%d]\n", height, width);

    return 0;
}