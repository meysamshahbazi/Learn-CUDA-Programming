
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}

__global__ void device_add(int *a, int *b,int *c)
{
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}



int main()
{
    int *a,*b,*c; // host vars

    int *d_a,*d_b,*d_c; // device copies of host vars

    int size = N*sizeof(int);

    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    // host_add(a,b,c);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    device_add<<<1,N>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size,hipMemcpyDeviceToHost);

    print_output(a,b,c);


    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    printf("\n");

    return 0;
}