
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

#define N 512

void host_add(int *a, int *b, int *c) {
	for(int idx=0;idx<N;idx++)
		c[idx] = a[idx] + b[idx];
}

//basically just fills the array with index.
void fill_array(int *data) {
	for(int idx=0;idx<N;idx++)
		data[idx] = idx;
}

void print_output(int *a, int *b, int*c) {
	for(int idx=0;idx<N;idx++)
		printf("\n %d + %d  = %d",  a[idx] , b[idx], c[idx]);
}

__global__ void device_add(int *a, int *b,int *c)
{
    int index = blockIdx.x*blockDim.x + threadIdx.x;
    c[index] = a[index] + b[index];
}



int main()
{
    int *a,*b,*c; // host vars

    int *d_a,*d_b,*d_c; // device copies of host vars

    int size = N*sizeof(int);

    a = (int *)malloc(size);
    fill_array(a);
    b = (int *)malloc(size);
    fill_array(b);
    c = (int *)malloc(size);

    hipMalloc((void **) &d_a, size);
    hipMalloc((void **) &d_b, size);
    hipMalloc((void **) &d_c, size);
    
    // host_add(a,b,c);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    device_add<<<16,32>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, size,hipMemcpyDeviceToHost);

    print_output(a,b,c);


    free(a);
    free(b);
    free(c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    printf("\n");

    return 0;
}