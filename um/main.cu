
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

__global__
void add(int n,float *x,float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i=index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

__global__
void init(int n, float *x,float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i =index; i <n; i+=stride)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}


int main(void)
{
    int N = 1<<20;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // for(int i=0; i<N; i++)
    // {
    //     x[i] = 1.0f;
    //     y[i] = 2.0f;
    // }

    int block_size = 256;
    int num_blocks = (N + block_size-1)/block_size;

    init<<<num_blocks,block_size>>>(N, x, y);
    
    add<<<num_blocks,block_size>>>(N,x,y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;
}



