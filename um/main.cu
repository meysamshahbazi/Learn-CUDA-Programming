#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>

__global__
void add(int n,float *x,float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i=index; i < n; i+=stride)
        y[i] = x[i] + y[i];
}

__global__
void init_(int n, float *x,float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x*gridDim.x;

    for (int i =index; i <n; i+=stride)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }
}

#define STRIDE_64K 65536

__global__
void init_align(int n, float *x,float *y)
{
    int lane_id = threadIdx.x & 31;
    // another way to say that
    // int lane_id = threadIdx.x % 32;

    size_t warp_id = (threadIdx.x + blockIdx.x*blockDim.x)>>5;

    size_t wapr_per_grid = (blockDim.x * gridDim.x)>>5;
    // size_t warp_total = 
    // TODO need to be completed !!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!!11

}


int main(void)
{
    int N = 1<<20;
    float *x, *y;
     
    int device = -1;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    for(int i=0; i<N; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    hipGetDevice(&device);
    hipMemPrefetchAsync(x, N*sizeof(float), device, NULL);
    hipMemPrefetchAsync(y, N*sizeof(float), device, NULL);

    int block_size = 256;
    int num_blocks = (N + block_size-1)/block_size;

    // init_<<<num_blocks,block_size>>>(N, x, y);
    
    add<<<num_blocks,block_size>>>(N,x,y);

    hipDeviceSynchronize();

    hipMemPrefetchAsync(y, N*sizeof(float), hipCpuDeviceId, NULL);
    
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x);
    hipFree(y);
    return 0;
}



