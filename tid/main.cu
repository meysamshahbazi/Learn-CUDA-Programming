
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_DIM_X 16
#define BLOCK_DIM_Y 16



////////////////////////////////////////////////////////////////////////////////
//! Compute reference data set matrix multiply on GPU
//! C = alpha * A * B + beta * C
//! @param A          matrix A as provided to device
//! @param B          matrix B as provided to device
//! @param C          matrix C as provided to device
//! @param N          height of matrix A and matrix C
//! @param M          width of matrix B and matrix C
//! @param K          width of matrix A and height of matrix B
//! @param alpha      scala value for matrix multiplication
//! @param beta       scala value for matrix summation with C
////////////////////////////////////////////////////////////////////////////////
__global__ void 
sgemm_gpu_kernel(const float *A, const float *B, float *C, int N, int M, int K, float alpha, float beta)
{
    int col = blockIdx.x*blockDim.x + threadIdx.x;
    int row = blockIdx.y*blockDim.y + threadIdx.y;

    float sum_ = 0;
    for(int i=0;i<K;i++)
    {
        sum_ +=A[ row*K + i ]*B[ i*K + col];
    }
    C[row*M+col] = alpha*sum_+beta*C[row*M+col];
}

__global__
void idx_print()
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int warp_idx = threadIdx.x/warpSize;
    int lane_idx = threadIdx.x & (warpSize -1);

    // if ( (lane_idx & (warpSize/2-1) ) == 0 ) 
        printf(" %5d\t%5d\t %2d\t%2d\n", idx, blockIdx.x, warp_idx, lane_idx);
}


int main(int argc, char* argv[])
{
    if (argc == 1) {
        puts("Please put Block Size and Thread Block Size..");
        puts("./tid [grid size] [block size]");
        puts("e.g.) ./tid 4 128");

        exit(1);
    }

    int grid_size = atoi(argv[1]);
    int block_size = atoi(argv[2]);

    puts("Thread, block, warp, lane");
    idx_print<<<grid_size,block_size>>>();
    hipDeviceSynchronize();

    return 0;
}


