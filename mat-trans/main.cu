
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 16
#define BLOCK_SIZE 4

__global__
void matrix_transpose_naive(int *input, int *output)
{
    int indexX = threadIdx.x + blockIdx.x*blockDim.x;
    int indexY = threadIdx.y + blockIdx.y*blockDim.y;
    int index = indexY*N + indexX;
    int transposedIndex = indexX*N + indexY;

    output[transposedIndex] = input[index];

    // another way :
    // output[index] = input[transposedIndex];

}

__global__
void matrix_trnspose_shared(int *input,int *output)
{
    // __shared__ int shared_mem [BLOCK_SIZE][BLOCK_SIZE];
    // use + 1 for no bank conflict?!
    __shared__ int shared_mem [BLOCK_SIZE][BLOCK_SIZE + 1];

    int indexX = threadIdx.x + blockIdx.x*blockDim.x;
    int indexY = threadIdx.y + blockIdx.y*blockDim.y;

    int tindexX = threadIdx.x + blockIdx.y * blockDim.x;
    int tindexY = threadIdx.y + blockIdx.x * blockDim.y;

    int localIndexX = threadIdx.x;
    int localIndexY = threadIdx.y;

    int index = indexY + N*indexX;
    int transpoesedIndex = tindexX + N*tindexY;


    shared_mem[localIndexX][localIndexY] = input[index];

    __syncthreads();

    output[transpoesedIndex] = shared_mem[localIndexY][localIndexX]; 
}


void fill_array(int *data) {
	for(int idx=0;idx<(N*N);idx++)
		data[idx] = idx;
}


void print_output(int *a, int *b) {
	printf("\n Original Matrix::\n");
	for(int idx=0;idx<(N*N);idx++) {
		if(idx%N == 0)
			printf("\n");
		printf(" %d ",  a[idx]);
	}
    
	printf("\n Transposed Matrix::\n");
	for(int idx=0;idx<(N*N);idx++) {
		if(idx%N == 0)
			printf("\n");
		printf(" %d ",  b[idx]);
	}
    printf("\n");
}




int main()
{
    int *a, *b;
    int *d_a, *d_b; // device copies of a, b, c

    int size = N * N *sizeof(int);

    a = (int *)malloc(size); fill_array(a);
	b = (int *)malloc(size);

    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_b, size);

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 blockSize(BLOCK_SIZE,BLOCK_SIZE,1);
    dim3 gridSize(N/BLOCK_SIZE,N/BLOCK_SIZE,1);

    // matrix_transpose_naive<<<gridSize,gridSize>>>(d_a,d_b);
    matrix_trnspose_shared<<<gridSize,gridSize>>>(d_a,d_b);

    hipMemcpy(b,d_b,size,hipMemcpyDeviceToHost);

    print_output(a,b);

    free(a);
    free(b);
    hipFree(d_a);
    hipFree(d_b);
    
    return 0;
}